#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

#include <THC/THC.h>
#include <THC/THCAtomics.cuh>
#include <THC/THCDeviceUtils.cuh>

#include <math.h>
#include <float.h>

#include <thrust/execution_policy.h>  // for certain cuda versions, this is where 'thrust::device' is

// #include <thrust/device_vector.h>
// #include <thrust/copy.h>
#include <thrust/extrema.h>
#include <hip/hip_runtime.h>

#include <iostream>

#include "hough_voting_cuda_utils.h"


#define THREADS_PER_BLOCK 512
#define VERTEX_CHANNELS 3 
#define MAX_ROI 64

#define PRINT(a) std::cout << #a << ": " << a << std::endl;

#define CUDA_1D_KERNEL_LOOP(i, n)                            \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; \
       i += blockDim.x * gridDim.x)


__global__ void compute_arrays_kernel(const int nthreads, const int* labelmap,
    int* arrays, int* array_size, const int height, const int width) 
{
  CUDA_1D_KERNEL_LOOP(index, nthreads) 
  {
    int n = index / (height * width);
    int mask = labelmap[index];
    if (mask > 0)
    {
      int size = atomicAdd(array_size + n, 1);
      int offset = n * height * width + size;
      arrays[offset] = index % (height * width);
    }
  }
}

__global__ void compute_hough_kernel(const int nthreads, float* hough_space, float* hough_data, 
    const float* vertmap, const float* extents, const float* meta_data, const int* arrays, const int* array_size, 
    const int* class_indexes, const int height, const int width, const float inlierThreshold, const int skip_pixels) 
{
  __shared__ float s_meta_data[9];

  CUDA_1D_KERNEL_LOOP(index, nthreads) 
  {
    if (threadIdx.x == 0)
    {
      for (int i = 0; i < 9; ++i)
      {
        s_meta_data[i] = meta_data[i]; 
      }
    }
    __syncthreads();

    // (cls, cx, cy) is an element in the hough space
    int n = index / (height * width);

    int cls = class_indexes[n];
    int pix = index % (height * width);
    int cx = pix % width;
    int cy = pix / width;
    int size = array_size[n];
    float distance = 0;
    float threshold = 0;

    for (int i = 0; i < size; i += skip_pixels)
    {
      int offset = n * height * width + i;
      int location = arrays[offset];  // H * W
      int x = location % width;
      int y = location / (width);

      // read the direction
      // offset = VERTEX_CHANNELS * cls + VERTEX_CHANNELS * num_classes * (y * width + x);
      offset = n * height * width * VERTEX_CHANNELS + (y * width + x) * VERTEX_CHANNELS;
      float u = vertmap[offset];
      float v = vertmap[offset + 1];
      float d = exp(vertmap[offset + 2]);

      // vote
      if (angle_distance(cx, cy, x, y, u, v) > inlierThreshold)
      {
        project_box(cls, extents, s_meta_data, d, 0.6, &threshold);
        float dx = fabsf(x - cx);
        float dy = fabsf(y - cy);
        if (dx < threshold && dy < threshold)
        {
          hough_space[index]++;
          distance += d;
        }
      }
    }

    if (hough_space[index] > 0)
    {
      distance /= hough_space[index];

      float bb_width = -1;
      float bb_height = -1;
      for (int i = 0; i < size; i += skip_pixels)
      {
        int offset = n * height * width + i;
        int location = arrays[offset];
        int x = location % width;
        int y = location / width;

        // read the direction
        // offset = VERTEX_CHANNELS * cls + VERTEX_CHANNELS * num_classes * (y * width + x);
        offset = n * height * width * VERTEX_CHANNELS + (y * width + x) * VERTEX_CHANNELS;
        float u = vertmap[offset];
        float v = vertmap[offset + 1];

        // vote
        if (angle_distance(cx, cy, x, y, u, v) > inlierThreshold)
        {
          project_box(cls, extents, s_meta_data, distance, 0.6, &threshold);
          float dx = fabsf(x - cx);
          float dy = fabsf(y - cy);
          if (dx > bb_width && dx < threshold && dy < threshold)
            bb_width = dx;
          if (dy > bb_height && dx < threshold && dy < threshold)
            bb_height = dy;
        }
      }

      int offset = n * height * width * 3 + 3 * (cy * width + cx);
      hough_data[offset] = distance;
      hough_data[offset + 1] = 2 * bb_height;
      hough_data[offset + 2] = 2 * bb_width;
    }
  }
}

__global__ void compute_rois_kernel(const int nthreads, float* top_box, float* top_pose, float* top_target, float* top_weight, int* top_domain,
    const float* extents, const float* meta_data, const float* hough_space, const float* hough_data, const int* max_indexes, const int* class_indexes,
    const int height, const int width) 
{
  __shared__ float s_f[4];

  CUDA_1D_KERNEL_LOOP(index, nthreads) 
  {
    if (threadIdx.x == 0)
    {
      s_f[0] = meta_data[0]; // fx
      s_f[1] = meta_data[4]; // fy
      s_f[2] = meta_data[2]; // px
      s_f[3] = meta_data[5]; // py
    }
    __syncthreads();

    float fx = s_f[0];
    float fy = s_f[1];
    float px = s_f[2];
    float py = s_f[3];

    float scale = 0.05;
    int max_index = max_indexes[index];
    float max_hs_idx = hough_space[max_index];
    int ind = max_index / (height * width);
    int cls = class_indexes[ind];
    int n = max_index % (height * width);
    int x = n % width;
    int y = n / width;

    float rx = (x - px) / fx;
    float ry = (y - py) / fy;

    int offset = ind * height * width * 3 + 3 * (y * width + x);
    float bb_distance = hough_data[offset];
    float bb_height = hough_data[offset + 1];
    float bb_width = hough_data[offset + 2];

    int roi_index = index; 
    top_box[roi_index * 7 + 0] = 0;
    top_box[roi_index * 7 + 1] = cls;
    top_box[roi_index * 7 + 2] = x - bb_width * (0.5 + scale);
    top_box[roi_index * 7 + 3] = y - bb_height * (0.5 + scale);
    top_box[roi_index * 7 + 4] = x + bb_width * (0.5 + scale);
    top_box[roi_index * 7 + 5] = y + bb_height * (0.5 + scale);
    top_box[roi_index * 7 + 6] = max_hs_idx;
    
    top_pose[roi_index * 7 + 0] = 1;
    top_pose[roi_index * 7 + 1] = 0;
    top_pose[roi_index * 7 + 2] = 0;
    top_pose[roi_index * 7 + 3] = 0;
    top_pose[roi_index * 7 + 4] = rx * bb_distance;
    top_pose[roi_index * 7 + 5] = ry * bb_distance;
    top_pose[roi_index * 7 + 6] = bb_distance;

  }
}

int HoughVotingForwardLaucher(
    const int* labels, const int* labelmap, const float* vertmap, const float* extents, const float* meta_data, const float* poses,
    const int batch_size, const int height, const int width, const int num_classes, 
    const int is_train, const float inlierThreshold, const int labelThreshold, const float votingThreshold, const float perThreshold, 
    const int skip_pixels, 
    float* top_box, float* top_pose, float* top_target, float* top_weight, int* top_domain, hipStream_t stream)
{
  const int kThreadsPerBlock = THREADS_PER_BLOCK;
  hipError_t err;

  const int N = batch_size;

  // step 1: compute a label index array for each instance
  int dims = N * height * width;
  int* arrays;// = arrays_vec.get();
  hipMalloc((void **)&arrays, dims * sizeof(int));
  hipMemset(arrays, 0, N * sizeof(int));

  int* array_sizes;// = array_sizes_vec.get();  
  hipMalloc((void **)&array_sizes, N * sizeof(int));
  hipMemset(array_sizes, 0, N * sizeof(int));

  int output_size = N * height * width;
  compute_arrays_kernel<<<(output_size + kThreadsPerBlock - 1) / kThreadsPerBlock,
                       kThreadsPerBlock, 0, stream>>>(
      output_size, labelmap, arrays, array_sizes, height, width);
  hipDeviceSynchronize();

  // DEBUG
  std::vector<int> array_sizes_host(N);
  hipMemcpy(&array_sizes_host[0], array_sizes, N * sizeof(int), hipMemcpyDeviceToHost);
  std::vector<int> labels_host(N);
  hipMemcpy(&labels_host[0], labels, N * sizeof(int), hipMemcpyDeviceToHost);
  std::vector<int> arrays_host(N*height*width);
  hipMemcpy(&arrays_host[0], arrays, N*height*width * sizeof(int), hipMemcpyDeviceToHost);
  // std::vector<float> meta_data_host(9);
  // hipMemcpy(&meta_data_host[0], meta_data, 9 * sizeof(float), hipMemcpyDeviceToHost);
  for (int n = 0; n < N; n++)
  {
    printf("Class %d) %d) (labels count: %d), sample value: %d\n", labels_host[n], n, array_sizes_host[n], arrays_host[n*height*width]); 
  }
  // for (int n = 0; n < 9; n++)
  // {
  //   printf("META: %.3f\n", meta_data_host[n]);
  // }
  // printf("\n");
  // 

  // step 2: compute the hough space
  float* hough_space; // = thrust::raw_pointer_cast(hough_space_vec.data());
  hipMalloc((void **)&hough_space, N * height * width * sizeof(float));
  if (hipMemset(hough_space, 0, N * height * width * sizeof(float)) != hipSuccess)
    fprintf(stderr, "reset error\n");

  float* hough_data; // = thrust::raw_pointer_cast(hough_data_vec.data());
  hipMalloc((void **)&hough_data, N * height * width * 3 * sizeof(float));
  if (hipMemset(hough_data, 0, N * height * width * 3 * sizeof(float)) != hipSuccess)
    fprintf(stderr, "reset error\n");

  output_size = N * height * width;
  compute_hough_kernel<<<(output_size + kThreadsPerBlock - 1) / kThreadsPerBlock,
                       kThreadsPerBlock, 0, stream>>>(
      output_size, hough_space, hough_data, vertmap, extents, meta_data,
      arrays, array_sizes, labels, height, width, inlierThreshold, skip_pixels);
  hipDeviceSynchronize();

  err = hipGetLastError();
  if(hipSuccess != err)
  {
    fprintf( stderr, "cudaCheckError() failed compute hough space: %s\n", hipGetErrorString( err ) );
    exit( -1 );
  }

  // step 3: find the maximum in hough space
  std::vector<int> max_indexes_host(N);
  for (int i = 0; i < N; i++)
  {
    float *hmax = thrust::max_element(thrust::device, hough_space + i * height * width, hough_space + (i+1) * height * width);
    max_indexes_host[i] = hmax - hough_space;
    printf("Max indexes %d) %d\n", i, max_indexes_host[i]);
  }

  int* max_indexes; 
  hipMalloc((void **)&max_indexes, N * sizeof(int));
  hipMemcpy(max_indexes, &max_indexes_host[0], N * sizeof(int), hipMemcpyHostToDevice);

  err = hipGetLastError();
  if(hipSuccess != err)
  {
    fprintf( stderr, "cudaCheckError() failed compute maximum: %s\n", hipGetErrorString( err ) );
    exit( -1 );
  }

  // step 4: compute outputs
  output_size = N;
  compute_rois_kernel<<<(output_size + kThreadsPerBlock - 1) / kThreadsPerBlock,
                       kThreadsPerBlock, 0, stream>>>(
      output_size, top_box, top_pose, top_target, top_weight, top_domain,
      extents, meta_data, hough_space, hough_data, max_indexes, labels,
      height, width);
  hipDeviceSynchronize();

  
  // err checking
  err = hipGetLastError();
  if(hipSuccess != err)
  {
    fprintf( stderr, "cudaCheckError() failed compute outputs: %s\n", hipGetErrorString( err ) );
    exit( -1 );
  }

  hipFree(arrays);
  hipFree(array_sizes);
  hipFree(hough_space);
  hipFree(hough_data);
  hipFree(max_indexes);

  return 1;
}


std::vector<at::Tensor> hough_voting_forward_cuda
(
    const at::Tensor& labels, const at::Tensor& masks, const at::Tensor& vertmap, const at::Tensor& extents, 
    const at::Tensor& meta_data, const at::Tensor& poses,
    const int num_classes, 
    const int is_train, const float inlierThreshold, const int labelThreshold, const float votingThreshold, const float perThreshold, 
    const int skip_pixels
) 
{
  // TODO:    at::Tensor& top_box, at::Tensor& top_pose, at::Tensor& top_target, at::Tensor& top_weight, at::Tensor& top_domain, at::Tensor& num_rois

  int batch_size = masks.size(0);
  int N = batch_size;
  int H = masks.size(1);
  int W = masks.size(2);

  printf("H: %d, H: %d, W: %d\n");

  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  // float tensors
  at::Tensor top_box = at::zeros({N, 7}, vertmap.options());
  at::Tensor top_pose = at::zeros({N, 7}, vertmap.options());
  at::Tensor top_target = at::zeros({N, 4*num_classes}, vertmap.options());
  at::Tensor top_weight = at::zeros({N, 4*num_classes}, vertmap.options());
  // int tensor
  at::Tensor top_domain = at::zeros({N}, labels.options());

  // resize_outputs(top_box, top_pose, top_target, top_weight, top_domain, max_allowed_rois, num_classes);
  // reset_outputs(top_box, top_pose, top_target, top_weight, top_domain);

  HoughVotingForwardLaucher(
        labels.contiguous().data<int>(), masks.contiguous().data<int>(), vertmap.contiguous().data<float>(), extents.contiguous().data<float>(), 
        meta_data.contiguous().data<float>(), poses.contiguous().data<float>(),
        N, H, W, num_classes,
        is_train, inlierThreshold, labelThreshold, votingThreshold, perThreshold, 
        skip_pixels,
        top_box.data<float>(), top_pose.data<float>(), top_target.data<float>(), top_weight.data<float>(), top_domain.data<int>(),
        stream
       );
  THCudaCheck(hipGetLastError());    

  // then resize outputs based on num_rois
  // printf("num_rois: %d\n", num_rois);
  // resize_outputs(top_box, top_pose, top_target, top_weight, top_domain, num_rois, num_classes);

  return {top_box, top_pose, top_target, top_weight, top_domain};
}
